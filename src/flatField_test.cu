#include "cute.h"
#include "flatField_test.h"
#include <stdio.h>
#include <tiffio.h>
#include <assert.h>
#include <iostream>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

//typedef unsigned short GreyscaleValue; //unsigned char for 8-bit and unsigned short for 16-bit tiff
//typedef double Real;

#include <flatFieldCorrect_cpu.h>
#include <flatFieldCorrect_gpu.h>

void flatFieldCorrect_cpu_test_16bit() {

	const char* lightFile = "./test-data/16-bit/MED_light.tif";
	const char* darkFile = "./test-data/16-bit/MED_dark.tif";
	const char* fileToCorrect = "./test-data/16-bit/lamb1_";
	int nImages = 1;

	float milli =0.0f;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	FieldImages fi(lightFile, darkFile);
	flatFieldCorrect_cpu<unsigned short, double>(fi, fileToCorrect,nImages);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);

	printf("testing execution of sequential flat field correction (16-bit): %.1f (ms)", milli);
	ASSERTM("execution failed", true);
}


void flatFieldCorrect_gpu_test_16bit() {

	const char* lightFile = "./test-data/16-bit/MED_light.tif";
	const char* darkFile = "./test-data/16-bit/MED_dark.tif";
	const char* fileToCorrect = "./test-data/16-bit/lamb1_";
	int nImages = 1;

	float milli =0.0f;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	FieldImages fi(lightFile, darkFile);
	flatFieldCorrect_gpu<unsigned short, double>(fi, fileToCorrect,nImages);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);

	printf("testing execution of parallel flat field correction (16-bit): %.1f (ms)", milli);
	ASSERTM("execution failed", true);
}

void flatFieldCorrect_cpu_test_8bit() {

	const char* lightFile = "./test-data/8-bit/light-median-gimp.tif";
	const char* darkFile = "./test-data/8-bit/dark-median-gimp.tif";
	const char* fileToCorrect = "./test-data/8-bit/sloth1_";
	int nImages = 1;

	float milli =0.0f;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	FieldImages fi(lightFile, darkFile);
	flatFieldCorrect_cpu<unsigned char, double>(fi, fileToCorrect, nImages);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);

	printf("testing execution of sequential flat field correction (8-bit): %.1f (ms)", milli);
	ASSERTM("execution failed", true);
}

void flatFieldCorrect_gpu_test_8bit() {

	const char* lightFile = "./test-data/8-bit/light-median-gimp.tif";
	const char* darkFile = "./test-data/8-bit/dark-median-gimp.tif";
	const char* fileToCorrect = "./test-data/8-bit/sloth1_";
	int nImages = 1;

	float milli =0.0f;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	FieldImages fi(lightFile, darkFile);
	flatFieldCorrect_gpu<unsigned char, double>(fi, fileToCorrect,nImages);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);

	printf("testing execution of parallel flat field correction (8-bit): %.1f (ms)", milli);
	ASSERTM("execution failed", true);
}

cute::suite make_suite_flatFieldCorrection(){
	cute::suite s;
	s.push_back(CUTE(flatFieldCorrect_cpu_test_16bit));
	s.push_back(CUTE(flatFieldCorrect_gpu_test_16bit));
	s.push_back(CUTE(flatFieldCorrect_cpu_test_8bit));
	s.push_back(CUTE(flatFieldCorrect_gpu_test_8bit));
	return s;
}



